#include <iostream>
#include <pcl/segmentation/extract_clusters.h>
#include <pcl/gpu/segmentation/gpu_extract_clusters.h>

#include <pcl/filters/voxel_grid.h>
#include <pcl/io/pcd_io.h>
#include <chrono>
#include <hip/hip_runtime.h>




int main(int argc, char** argv){
    std::cout<<"..........comparing gpu and cpu clustering..........."<<std::endl;

    pcl::PCDReader reader;
    pcl::PointCloud<pcl::PointXYZ>::Ptr in_cloud(new pcl::PointCloud<pcl::PointXYZ>());

    reader.read("../table_scene_lms400.pcd", *in_cloud);
    
    std::cout<<"\n\n\n\tPoint cloud read with size: "<<in_cloud->size()<<"\n\n"<<std::endl;


    // filter with voxelgrid approach
    pcl::VoxelGrid<pcl::PointXYZ> vg;
    pcl::PointCloud<pcl::PointXYZ>::Ptr out_cloud(new pcl::PointCloud<pcl::PointXYZ>);
    vg.setInputCloud(in_cloud);
    vg.setLeafSize(0.01f,0.01f,0.01f);

    auto start_filter=std::chrono::steady_clock::now();
    vg.filter(*out_cloud);
    auto end_filter=std::chrono::steady_clock::now();


    std::cout<<"\t\t\tthe new cloud size is "<<out_cloud->size()<<std::endl;
    std::cout<<"\t\t\ttime took for filtering was: "
    << std::chrono::duration_cast<std::chrono::milliseconds>(end_filter-start_filter).count()<<std::endl;
    std::cout<<"\t\t\tThe 400k points are fed to cpu"<<std::endl;

    // setting the KDtree and cluster in cpu
    pcl::search::KdTree<pcl::PointXYZ>::Ptr kdtree(new pcl::search::KdTree<pcl::PointXYZ>);
    kdtree->setInputCloud(in_cloud);
    std::vector<pcl::PointIndices> cluster_indices;

    boost::shared_ptr<pcl::EuclideanClusterExtraction<pcl::PointXYZ>> ec(new pcl::EuclideanClusterExtraction<pcl::PointXYZ>);

    ec->setClusterTolerance(0.02);
    ec->setMinClusterSize(100);
    ec->setMaxClusterSize(25000);
    ec->setSearchMethod(kdtree);
    ec->setInputCloud(in_cloud);
    
    
    auto start_clustering_cpu=std::chrono::steady_clock::now();
    ec->extract(cluster_indices);
    auto end_clustering_cpu=std::chrono::steady_clock::now();

    std::cout<<"\n\n\tThe time took for the cpu version was: "<<
    std::chrono::duration_cast<std::chrono::milliseconds>(end_clustering_cpu-start_clustering_cpu).count()<<std::endl;
    std::cout<<"\tThe size of the cluster indices is: "<<cluster_indices.size()<<std::endl;

    // now for the gpu version
    pcl::gpu::Octree::Ptr octree_device(new pcl::gpu::Octree);
    
    pcl::gpu::Octree::PointCloud gpu_pointcloud;
    pcl::gpu::EuclideanClusterExtraction< pcl::PointXYZ>::PointCloudHostPtr host_in_cloud = in_cloud;


    gpu_pointcloud.upload(in_cloud->points);

    std::cout<<"\n\n\n\tsize of the gpu point cloud: "<<gpu_pointcloud.size()<<std::endl;
    octree_device->setCloud(gpu_pointcloud);
    pcl::gpu::EuclideanClusterExtraction<pcl::PointXYZ> gpu_ec;
    std::vector<pcl::PointIndices> pci;
    
    
    gpu_ec.setClusterTolerance(0.02);
    gpu_ec.setMinClusterSize(100);
    gpu_ec.setMaxClusterSize(25000);
    gpu_ec.setSearchMethod(octree_device);
    gpu_ec.setHostCloud(in_cloud);  
    
    
    auto gpu_start=std::chrono::steady_clock::now();  
    gpu_ec.extract(pci);
    auto gpu_end=std::chrono::steady_clock::now();




    
    std::cout<<"\tThe time took for gpu version was: "<<
    std::chrono::duration_cast<std::chrono::milliseconds>(gpu_end - gpu_start).count()<<std::endl;
    std::cout<<"\tThe size of cluster inidices is: "<<pci.size()<<" number of clusters"<<std::endl;



    std::cout<<"\n\n\n\tExiting the program"<<std::endl;



    return 0;
}